
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 32
#define FULL_MASK 0xffffffff

__constant__ float filter[3 * 3];


__device__ void full_load(const float *input, float *s_input, int rows, int cols, int padding, unsigned total_padding, int tile_size, int tid_x, int tid_y) {
    int overall_x = tid_x - padding;
    int overall_y = tid_y - padding;
    unsigned wid = (threadIdx.y * BLOCK_SIZE + threadIdx.x) / warpSize;
    s_input[threadIdx.y * tile_size + threadIdx.x] = (overall_x >= 0 && overall_x < cols && overall_y >= 0 && overall_y < rows) ? input[overall_y * cols + overall_x] : 0.f;
   
    unsigned num_warps = total_padding + (tile_size + 2 - 1) / 2;  // ceiling division
    if (wid < total_padding) {
        overall_y += BLOCK_SIZE;
        if ((BLOCK_SIZE + threadIdx.y) < tile_size && threadIdx.x < tile_size) {
            s_input[(BLOCK_SIZE + threadIdx.y) * tile_size + threadIdx.x] = (overall_x >= 0 && overall_x < cols && overall_y >= 0 && overall_y < rows) ? \
            input[overall_y * cols + overall_x] : 0.f;
        }
    }
    else if (wid < num_warps) {
        unsigned shared_y = threadIdx.x / 16 + 2 * (wid - total_padding);
        unsigned shared_x = BLOCK_SIZE + threadIdx.x % total_padding;
        if (shared_x < tile_size && shared_y < tile_size) {
            overall_y = blockIdx.y * blockDim.y + shared_y - padding;
            overall_x = blockIdx.x * blockDim.x + shared_x - padding;
            s_input[shared_y * tile_size + shared_x] = \
            (overall_x >= 0 && overall_x < cols && overall_y >= 0 && overall_y < rows) ? input[overall_y * cols + overall_x] : 0.f;
        }
    }
}


__global__ void conv_2d_shfl(const float *input, float *out, int rows, int cols, int filter_size) {
    int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;

    int padding = filter_size / 2;
    unsigned total_padding = padding * 2;
    int tile_size = BLOCK_SIZE + padding * 2;
    float val = 0.f;

    extern __shared__ float s_input[];

    // copy global input into shared input
    for (int tile_row = 0; tile_row < tile_size / BLOCK_SIZE; ++tile_row) {
            int tile_y = threadIdx.y + tile_row * BLOCK_SIZE;
            int overall_y = tid_y - padding + tile_row * BLOCK_SIZE;
        for (int tile_col = 0; tile_col < tile_size / BLOCK_SIZE; ++tile_col) {
            // three regions - inside array, outside array inside padding, outside padding
            int tile_x = threadIdx.x + tile_col * BLOCK_SIZE;
            if (tile_x >= tile_size || tile_y >= tile_size) continue;  // outside padding/(array where thread should read from)

            // everything else will be inside array or padding
            int overall_x = tid_x - padding + tile_col * BLOCK_SIZE;
            s_input[tile_y * tile_size + tile_x] = (overall_x >= 0 && overall_x < cols && overall_y >= 0 && overall_y < rows) ? input[overall_y * cols + overall_x] : 0.f;
        }
    }
    __syncthreads();

    bool inside_array = tid_x >= 0 && tid_x < cols && tid_y >= 0 && tid_y < rows;
    float2 next_vals;
    for (int fid_y = 0; fid_y < filter_size; ++fid_y) {
        int idx_y = threadIdx.y + fid_y;
        float arr_val = s_input[idx_y * tile_size + threadIdx.x];  // initial row load
        if (threadIdx.x == 31) next_vals = reinterpret_cast<float2*>(s_input)[32];  // vectorized load by last thread
        val += inside_array ? arr_val * filter[fid_y * filter_size] : 0.f;
        arr_val = __shfl_down_sync(FULL_MASK, arr_val, 1);  // loading from threads above
        next_vals.x = __shfl_down_sync(FULL_MASK, arr_val, 1);
        for (int fid_x = 1; fid_x < filter_size; ++fid_x) {
            val += inside_array ? arr_val * filter[fid_y * filter_size + fid_x] : 0.f;
            arr_val = next_vals.x;
        }
    }

    if (inside_array)
        out[tid_y * cols + tid_x] = val;
}


void fill_array(float *arr, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        arr[i] = 1.f;
    }
}


void print_matrix(float *arr, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%.2f ", arr[i * cols + j]);
        }
        printf("\n");
    }
}


int main() {
    // Event for timing

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    int rows = 2048, cols = 2048;
    int filter_size = 3;
    float *img, *h_filter, *out;
    float *d_img, *d_out;

    // allocate memory
    hipHostMalloc((void **)&img, rows * cols * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_filter, filter_size * filter_size * sizeof(float), hipHostMallocDefault);
    out = (float *)malloc(rows * cols * sizeof(float));

    fill_array(img, rows, cols);
    fill_array(h_filter, filter_size, filter_size);

    hipMalloc((void **)&d_img, rows * cols * sizeof(float));
    hipMalloc((void **)&d_out, rows * cols * sizeof(float));

    // copy initialized arrays to device memory
    hipMemcpy(d_img, img, rows * cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(filter), h_filter, filter_size * filter_size * sizeof(float));

    //kernel call
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 dimGrid((rows + BLOCK_SIZE - 1) / BLOCK_SIZE, (cols + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
    int tile_size = BLOCK_SIZE + (filter_size / 2) * 2;
    conv_2d_shfl<<<dimGrid, dimBlock, tile_size * tile_size * sizeof(float), 0>>>(d_img, d_out, rows, cols, filter_size);
    //cudaDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time_ = 0.f;
    hipEventElapsedTime(&time_, start, stop);
    printf("Shuffle elapsed time (ms): %.3f \n", time_);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(out, d_out, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
    // print_matrix(out, rows, cols);

    // frees
    hipFree(d_img);
    hipFree(d_out);
    hipHostFree(img);
    hipHostFree(h_filter);
    free(out);

    return 0;
}

