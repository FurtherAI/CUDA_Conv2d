
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 32
#define MAX_FILTER 8
#define FULL_MASK 0xffffffff

__constant__ float filter[MAX_FILTER * MAX_FILTER];


__global__ void conv_2d_tex(hipTextureObject_t texObj, float *out, int rows, int cols, int filter_size) {
    int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;

    int padding = filter_size / 2;

    float val = 0.f;
    for (int fid_y = 0; fid_y < filter_size; ++fid_y) {
        for (int fid_x = 0; fid_x < filter_size; ++fid_x) {
            int idx_x = tid_x - padding + fid_x;
            int idx_y = tid_y - padding + fid_y;
            val += (tid_x >= 0 && tid_x < cols && tid_y >= 0 && tid_y < rows) ? tex2D<float>(texObj, (float)idx_x, (float)idx_y) * filter[fid_y * filter_size + fid_x] : 0.f;
        }
    }

    if (tid_x >= 0 && tid_x < cols && tid_y >= 0 && tid_y < rows)
        out[tid_y * cols + tid_x] = val;
}


void fill_array(float *arr, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        arr[i] = 1.f;
    }
}


void print_matrix(float *arr, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%.2f ", arr[i * cols + j]);
        }
        printf("\n");
    }
}


int main() {
    // Event for timing

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    int rows = 2048, cols = 2048;
    int filter_size = 3;
    float *img, *h_filter, *out, *d_out;

    // allocate cudaArray (on devic) which holds the image
    hipArray_t d_img;
    hipChannelFormatDesc c_format = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipMallocArray(&d_img, &c_format, cols, rows);

    // allocate memory for host image, host filter, host output and device output
    hipHostMalloc((void **)&img, rows * cols * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_filter, filter_size * filter_size * sizeof(float), hipHostMallocDefault);
    out = (float *)malloc(rows * cols * sizeof(float));
    hipMalloc((void **)&d_out, rows * cols * sizeof(float));

    // fill arrays with ones (easy to check for correct output)
    fill_array(img, rows, cols);
    fill_array(h_filter, filter_size, filter_size);

    // copy filter to constant memory and image to cuda array for texture memory
    hipMemcpyToSymbol(HIP_SYMBOL(filter), h_filter, filter_size * filter_size * sizeof(float));
    hipMemcpy2DToArray(d_img, 0, 0, img, cols * sizeof(float), cols * sizeof(float), rows, hipMemcpyHostToDevice);

    // create resource description which holds array
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = d_img;

    // create texture description which defines access behavior
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // create texture object
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    //kernel call
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 dimGrid((rows + BLOCK_SIZE - 1) / BLOCK_SIZE, (cols + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
    int tile_size = BLOCK_SIZE + (filter_size / 2) * 2;
    conv_2d_tex<<<dimGrid, dimBlock, tile_size * tile_size * sizeof(float), 0>>>(texObj, d_out, rows, cols, filter_size);
    // cudaDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time_ = 0.f;
    hipEventElapsedTime(&time_, start, stop);
    printf("Texture elapsed time (ms): %.3f \n", time_);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(out, d_out, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
    // print_matrix(out, rows, cols);

    hipDestroyTextureObject(texObj);

    // frees
    hipFreeArray(d_img);
    hipFree(d_out);
    hipHostFree(img);
    hipHostFree(h_filter);
    free(out);

    return 0;
}

